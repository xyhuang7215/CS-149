#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

// helper functions to execute exclusive_scan
__global__ 
void up_sweep(int * array, int offset) {
    int thid = threadIdx.x + blockDim.x * blockIdx.x;
    int ai = offset * (2 * thid + 1) - 1;
    int bi = offset * (2 * thid + 2) - 1;
    array[bi] += array[ai];
}


__global__
void down_sweep(int * array, int offset) {
    int thid = threadIdx.x + blockDim.x * blockIdx.x;
    int ai = offset * (2 * thid + 1) - 1;
    int bi = offset * (2 * thid + 2) - 1;
    int t = array[ai];
    array[ai] = array[bi];
    array[bi] += t;
}

__global__ 
void clear_last(int * array, int N) {
    array[N - 1] = 0;
}

// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep input
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.

    const int blockSize = 512;

    // up sweep
    int offset = 1;
    int d = N / 2;
    while (d > blockSize) {
        int gridSize = d / blockSize;
        up_sweep<<<gridSize, blockSize>>>(result, offset);
        offset *= 2;
        d /= 2;
    }
    while (d > 0) {
        up_sweep<<<1, d>>>(result, offset);
        offset *= 2;
        d /= 2;
    }

    // clear the last element
    clear_last<<<1, 1>>>(result, N);

    // down sweep
    d = 1;
    while (d <= blockSize) {
        offset /= 2;
        down_sweep<<<1, d>>>(result, offset);
        d *= 2;
    }
    while (d < N) {
        offset /= 2;
        int gridSize = d / blockSize;
        down_sweep<<<gridSize, blockSize>>>(result, offset);
        d *= 2;
    }
}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    // int N = end - inarray;

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.

    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, rounded_length, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_result);
    hipFree(device_input);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// helper functions to execute find_repeats
__global__
void consecutive_cmp(int *input, int *output, int length) {
    int thid = threadIdx.x + blockDim.x * blockIdx.x;
    if (thid < length - 1) {
        output[thid] = input[thid] == input[thid + 1] ? 1 : 0;
    }
}

__global__
void consecutive_sub(int *input, int *output, int length) {
    int thid = threadIdx.x + blockDim.x * blockIdx.x;
    if (thid < length - 1) {
        output[thid] = input[thid + 1] - input[thid];
    }
}

__global__
void collect_repeats(int *prefix_sum, int *repeats_indicate, int *output, int length) {
    int thid = threadIdx.x + blockDim.x * blockIdx.x;
    if (thid < length - 1) {
        if (repeats_indicate[thid] == 1) {
            output[prefix_sum[thid]] = thid;
        }
    }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output, int* temp) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
    int blockSize = 512;
    int rounded_length = nextPow2(length);
    if (length <= blockSize) {
        consecutive_cmp<<<1, length - 1>>>(device_input, temp, length);
        exclusive_scan(temp, rounded_length, temp);
        consecutive_sub<<<1, length - 1>>>(temp, device_input, length);
        collect_repeats<<<1, length - 1>>>(temp, device_input, device_output, length);
    } else {
        int gridSize = (length + blockSize - 1) / blockSize;
        consecutive_cmp<<<gridSize, blockSize>>>(device_input, temp, length);
        exclusive_scan(temp, rounded_length, temp);
        consecutive_sub<<<gridSize, blockSize>>>(temp, device_input, length);
        collect_repeats<<<gridSize, blockSize>>>(temp, device_input, device_output, length);
    }
    int * repeats_num = new int;
    hipMemcpy(repeats_num, temp + length - 1, sizeof(int), hipMemcpyDeviceToHost);    
    return *repeats_num; 
}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int *temp;
    int rounded_length = nextPow2(length);

    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMalloc((void **)&temp, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_output, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output, temp);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);
    hipFree(temp);

    float duration = endTime - startTime; 
    return duration;
}


void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
